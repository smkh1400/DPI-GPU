
#include <hip/hip_runtime.h>
// #include <cuda_runtime.h>
// #include <pcap.h>
// #include <stdlib.h>
// #include <stdio.h>
// #include <string.h>
// #include <endian.h>
// #include <unistd.h>
// #include <dirent.h>

// #include "rulesGraph.cuh"
// #include "header.h"
// #include "gputimer.cuh"
// #include "rules.cuh"
// #include "config/config.h"

// #define CHECK_CUDA_ERROR(fun)                                                   \
// {                                                                               \
//     cudaError_t err = fun;                                                      \
//     if(err != cudaSuccess) {                                                    \
//         printf("CUDA at %s:%d: %s\n", __FUNCTION__, __LINE__ , cudaGetErrorString(err));           \
//         return -1;                                                               \
//     }                                                                           \
// }

// #define ALIGN_ADDRESS(addr, struct, alignedAddr)                {                                                                                   \
//                                                                     size_t alignment = alignof(struct);                                             \
//                                                                     uintptr_t ptr = (uintptr_t) (addr);                                             \
//                                                                     void* alignedAddress = (void*) ((ptr + (alignment - 1)) & ~(alignment - 1)) ;   \
//                                                                     alignedAddr = (struct*) alignedAddress;                                         \
//                                                                 } 

// #define HD_MODULE(timerHD, h_packetsMetadata, d_packetsMetadata,  h_packetsMemPool, d_packetsMemPool, counter, stream, durationHD, totalHDDuration, verbose, packetSize, streamName)                            \
// {                                                                                                                                                                                                               \
//     if (Configfields::isTimerSet) timerHD.start();                                                                                                                                                              \
//     CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMemPool, (void*) h_packetsMemPool, packetSize, cudaMemcpyHostToDevice, stream));                                                                          \
//     CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMetadata, (void*) h_packetsMetadata, counter * sizeof(PacketMetadata), cudaMemcpyHostToDevice, stream));                                                  \
//     if (Configfields::isTimerSet) timerHD.end();                                                                                                                                                                \
//     if (Configfields::isTimerSet) durationHD = timerHD.elapsed();                                                                                                                                               \
//     if (Configfields::isTimerSet) totalHDDuration += durationHD;                                                                                                                                                \
//     if(verbose) printf("[%s] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Host To Device \n", streamName, (counter), (packetSize)/(_GB_), (counter * sizeof(PacketMetadata)) / (_GB_));   \
//     if(verbose && Configfields::isTimerSet) printf("[%s]\t| DurationHD : %lf ms\n", streamName, durationHD);                                                                                                          \
//     if(verbose && Configfields::isTimerSet) printf("[%s]\t| BandwidthHD : %lf Gb/s\n", streamName, ((packetSize + counter * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationHD));                               \
// }           

// #define KERNEL_MODULE(timerKernel, counter, threadPerBlock, stream, d_packetsMetadata, d_packetsMemPool, d_trie, durationKernel, totalKernelDuration, verbose, packetSize, streamName)  \
// {                                                                                                                                                                                       \
//     if (Configfields::isTimerSet) timerKernel.start();                                                                                                                                  \
//     performProcess<<<((counter + threadPerBlock - 1)/threadPerBlock), threadPerBlock, 0, stream>>>(d_packetsMetadata, d_packetsMemPool, counter, d_trie);                               \
//     if (Configfields::isTimerSet) timerKernel.end();                                                                                                                                    \
//     if (Configfields::isTimerSet) durationKernel = timerKernel.elapsed();                                                                                                               \
//     if (Configfields::isTimerSet) totalKernelDuration += durationKernel;                                                                                                                \
//     if(verbose) printf("[%s] -  RuleGraph Was Processed For %d Threads Per Block\n", streamName, threadPerBlock);                                                                       \
//     if(verbose) printf("[%s] - %ld Packets (%.3lf GB) Processed On GPU\n", streamName, counter, ((packetSize) * 1.0)/(_GB_));                                                           \
//     if(verbose  && Configfields::isTimerSet) printf("[%s]\t| DurationKernel : %lf ms\n", streamName, durationKernel);                                                                   \
//     if(verbose  && Configfields::isTimerSet) printf("[%s]\t| BandwidthKernel : %lf Gb/s\n", streamName, ((packetSize) * 1000.0 * 8.0)/(_GB_*durationKernel));                           \
// }                                                                                                                                           

// #define DH_MODULE(timerDH, h_packetsMetadata, d_packetsMetadata, h_packetsMemPool, d_packetsMemPool, counter, stream, durationDH, totalDHDuration, verbose, packetSize, streamName)                                  \
// {                                                                                                                                                                                                                   \
//     if (Configfields::isTimerSet) timerDH.start();                                                                                                                                                                  \
//     CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMetadata, (void*) d_packetsMetadata, counter * sizeof(PacketMetadata), cudaMemcpyDeviceToHost, stream));                                                      \
//     CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMemPool, (void*) d_packetsMemPool, packetSize, cudaMemcpyDeviceToHost, stream));                                                                              \
//     if (Configfields::isTimerSet) timerDH.end();                                                                                                                                                                    \
//     if (Configfields::isTimerSet) durationDH = timerDH.elapsed();                                                                                                                                                   \
//     if (Configfields::isTimerSet) totalDHDuration += durationDH;                                                                                                                                                    \
//     if(verbose) printf("[%s] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Device to Host\n", streamName, counter, (packetSize)/(_GB_), (counter * sizeof(PacketMetadata)) / (_GB_));          \
//     if(verbose && Configfields::isTimerSet) printf("[%s]\t| DurationDH : %lf ms\n", streamName, durationDH);                                                                                                        \
//     if(verbose && Configfields::isTimerSet) printf("[%s]\t| BandwidthDH : %lf Gb/s\n", streamName, ((packetSize + counter * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationDH));                             \
// }

// #define _GB_                (1024.0*1024.0*1024.0)             
// #define _MB_                (1024.0*1024.0)             
// #define _KB_                (1024.0)             

// #define HOST_RAM_SIZE       (60.0 * _GB_)    
// #define DEVICE_RAM_SIZE     (21.0 * _GB_)

// #define MIN(x, y)           ((x) < (y) ? (x) : (y))

// #define DEFAULT_PACKET_BUFFER_CHUNK_SIZE    (196608*8)
// #define PACKET_BUFFER_CHUNK_SIZE            ((Configfields::chunkCountLimit != CONFIG_FIELD_INT_NOT_SET_VAL) ? Configfields::chunkCountLimit : DEFAULT_PACKET_BUFFER_CHUNK_SIZE)

// #define RULE_TRIE_SIZE                      (sizeof(RuleTrie))
// #define PACKETS_INFO_SIZE                   (PACKET_BUFFER_CHUNK_SIZE*sizeof(PacketInfo))
// #define PACKETS_METADATA_SIZE               (PACKET_BUFFER_CHUNK_SIZE*sizeof(PacketMetadata))
// #define PACKETS_MEMPOOL_SIZE                (MIN(DEVICE_RAM_SIZE, HOST_RAM_SIZE) - (PACKETS_METADATA_SIZE + RULE_TRIE_SIZE))

// __global__ void performProcess(PacketMetadata* packetsMetadata, uint8_t* packetsMempool, size_t packetCount, RuleTrie* trie) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= packetCount) return;

//     PacketMetadata md;     
//     md = packetsMetadata[idx];     

//     HeaderBuffer h(packetsMempool + (md.packetOffset + sizeof(PacketInfo)), md.packetLen);

//     PacketInfo* info;
//     ALIGN_ADDRESS(packetsMempool+md.packetOffset, PacketInfo, info);

//     trie->processTrie(&h);
//     info->ruleId = h.ruleId;
// }

// static int readPacketOfflineMode(PacketMetadata* packetsMetadata, uint8_t* packetsMempool ,pcap_t* handle, size_t* counter, size_t* packetSize, double* startTime) {
//     size_t packetOffset = 0;
//     size_t packetCounter = 0;
//     int result;
//     static const u_char *packet;
//     static struct pcap_pkthdr *header;
//     double timeStamp;


//     do {
//         if(packet != NULL && header != NULL) {
//             // if(header->caplen != header->len) continue;

//             timeStamp = (double)(header->ts.tv_sec) + (double)((header->ts.tv_usec*1.0) / 1e6f);

//             if((Configfields::chunkTimeLimit != CONFIG_FIELD_DOUBLE_NOT_SET_VAL) && (timeStamp - *startTime > Configfields::chunkTimeLimit / 3)) break;                                              // Time Limit

//             if((packetCounter >= PACKET_BUFFER_CHUNK_SIZE / 3)) break;                                                     // Count Limit
//             PacketMetadata md = {.packetOffset = packetOffset, .packetLen = header->caplen};
//             packetsMetadata[packetCounter] = md;

//             if(md.packetOffset+md.packetLen+sizeof(PacketInfo) >= PACKETS_MEMPOOL_SIZE / 3) break;                          // Mempool Limit
//             memcpy(packetsMempool+md.packetOffset+sizeof(PacketInfo), packet, md.packetLen);

//             packetCounter += 1;
//             packetOffset += md.packetLen+sizeof(PacketInfo);
//         }
//     }
//     while(((result = (pcap_next_ex(handle, &header, &packet))) >= 0));

//     *startTime = timeStamp;
//     *packetSize = packetOffset;
//     *counter = packetCounter;

//     return result;
// }

// static inline pcap_t* openPcapFile(const char* pcapFilePath) {
//     char errBuf[PCAP_ERRBUF_SIZE];
//     pcap_t* handle = (pcap_t*) malloc(sizeof(pcap_t*));
    
//     handle = pcap_open_offline(pcapFilePath, errBuf);

//     return handle;
// } 

// static double findFirstTimeStamp(const char* pcapFilePath) {
//     pcap_t* handle = openPcapFile(pcapFilePath);
//     struct pcap_pkthdr *header;
//     const u_char *packet;
//     int result;

//     if ((result = (pcap_next_ex(handle, &header, &packet))) < 0) return (int) result;
//     double firstTime = (double) header->ts.tv_sec + (double) ((header->ts.tv_usec*1.0) / 1e6f);

//     pcap_close(handle);

//     return firstTime;
// }

// static inline bool hasPcapExtension(const char* filename) {
//     const char* ext = strrchr(filename, '.');
//     return (ext != NULL) && (strcmp(ext, ".pcap") == 0);
// }

// static int processPcapFile(const char* pcapFilePath, bool verbose) {


//     if(!hasPcapExtension(pcapFilePath)) {
//         printf("Invalid Extension, Excepted .pcap\n");
//         return -1;
//     }

//     pcap_t* handle;
//     handle = openPcapFile(pcapFilePath);

//     if(handle == NULL) {
//         printf("Unable To Open Pcap File : %s\n", pcapFilePath);
//         return -1;
//     } 

//     if(verbose) printf("Pcap File %s Opened\n", pcapFilePath);
    
//     size_t pcapFileSize;
//     {
//         FILE* fd = fopen(pcapFilePath, "r");

//         fseek(fd, 0, SEEK_END);
//         pcapFileSize = ftell(fd);
//         fseek(fd, 0, SEEK_SET);

//         fclose(fd);
//     }  


//     PacketMetadata* h_packetsMetadataPing;
//     PacketMetadata* h_packetsMetadataPong;
//     PacketMetadata* h_packetsMetadataPang;
//     PacketMetadata* d_packetsMetadataPing;
//     PacketMetadata* d_packetsMetadataPong;
//     PacketMetadata* d_packetsMetadataPang;

//     uint8_t* d_packetsMemPoolPing;
//     uint8_t* d_packetsMemPoolPong;
//     uint8_t* d_packetsMemPoolPang;
//     uint8_t* h_packetsMemPoolPing;
//     uint8_t* h_packetsMemPoolPong;
//     uint8_t* h_packetsMemPoolPang;

//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMetadataPing, PACKETS_METADATA_SIZE / 3, cudaHostAllocDefault));
//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMetadataPong, PACKETS_METADATA_SIZE / 3, cudaHostAllocDefault));
//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMetadataPang, PACKETS_METADATA_SIZE / 3, cudaHostAllocDefault));
//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMemPoolPing, PACKETS_MEMPOOL_SIZE / 3, cudaHostAllocDefault));
//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMemPoolPong, PACKETS_MEMPOOL_SIZE / 3, cudaHostAllocDefault));
//     CHECK_CUDA_ERROR(cudaHostAlloc((void**) &h_packetsMemPoolPang, PACKETS_MEMPOOL_SIZE / 3, cudaHostAllocDefault));

//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMetadataPing, PACKETS_METADATA_SIZE / 3));
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMetadataPong, PACKETS_METADATA_SIZE / 3));
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMetadataPang, PACKETS_METADATA_SIZE / 3));
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMemPoolPing, PACKETS_MEMPOOL_SIZE / 3));
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMemPoolPong, PACKETS_MEMPOOL_SIZE / 3));
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_packetsMemPoolPang, PACKETS_MEMPOOL_SIZE / 3));

//     RuleTrie* d_trie;
//     CHECK_CUDA_ERROR(cudaMalloc((void**) &d_trie, RULE_TRIE_SIZE)); 
    

//     if(h_packetsMetadataPing == NULL || h_packetsMetadataPong == NULL || h_packetsMetadataPang == NULL || h_packetsMemPoolPing == NULL || h_packetsMemPoolPong == NULL || h_packetsMemPoolPang == NULL) {
//         printf("Unable to allocate Mempool or Metadata\n");
//         return -1;
//     }

//     CHECK_CUDA_ERROR(cudaThreadSetLimit(cudaLimitStackSize, 10*1024));

//     registerRules<<<1,1>>>(d_trie);
//     CHECK_CUDA_ERROR(cudaDeviceSynchronize());
//     if(verbose) printf(">> RuleGraph Was Registered On Device\n");

//     size_t counterPing;
//     size_t counterPong;
//     size_t counterPang;
//     size_t packetSizePing;
//     size_t packetSizePong;
//     size_t packetSizePang;
//     size_t chunkCounter = 0;

//     size_t totalCounter = 0;
//     size_t totalPacketSize = 0;
//     size_t totalHDPacketSize = 0;
//     size_t totalDHPacketSize = 0;

//     double totalHDDuration = 0;
//     double totalDHDuration = 0;
//     double totalKernelDuration = 0;

//     int ruleCount[Rule_Count] = {0};

//     cudaStream_t pingStream;
//     cudaStream_t pongStream;
//     cudaStream_t pangStream;

//     CHECK_CUDA_ERROR(cudaStreamCreate(&pingStream));
//     CHECK_CUDA_ERROR(cudaStreamCreate(&pongStream));
//     CHECK_CUDA_ERROR(cudaStreamCreate(&pangStream));

//     float durationChunk;
//     GPUTimer timerChunk(0);
//     double totalDuration = 0;

//     float durationDHPing;
//     GPUTimer timerDHPing(pingStream);

//     float durationDHPong;
//     GPUTimer timerDHPong(pongStream);

//     float durationDHPang;
//     GPUTimer timerDHPang(pangStream);

//     float durationHDPing;
//     GPUTimer timerHDPing(pingStream);

//     float durationHDPong;
//     GPUTimer timerHDPong(pongStream);

//     float durationHDPang;
//     GPUTimer timerHDPang(pangStream);

//     float durationKernelPing;
//     GPUTimer timerKernelPing(pingStream);

//     float durationKernelPong;
//     GPUTimer timerKernelPong(pongStream);

//     float durationKernelPang;
//     GPUTimer timerKernelPang(pangStream);

//     double startTime = findFirstTimeStamp(pcapFilePath);
//     int result = 0;

//     FILE* resultFd = fopen("/home/hamed/RuleGraph/GPU/tuning/result.csv", "a");
//     if(resultFd == NULL)    {
//         printf("Failed\n");
//         return -1;
//     }

//     while (1) {

//         if(verbose) printf(">> Chunk %ld Started\n", chunkCounter+1);

//         if(result != -2)
//         {
//             //ping
//             if (Configfields::readPacketMode.compare("offline") == 0)                // TODO 
//                 result = readPacketOfflineMode(h_packetsMetadataPing, h_packetsMemPoolPing, handle, &counterPing, &packetSizePing, &startTime);
//             else {
//                 printf("Invalid Read Mode in Config file\n");
//                 return -1;
//             }

//             if(result < 0 && result != -2 && verbose) {
//                 printf("Something went wrong in reading packets(%d)\n", result);
//                 printf("The Error was : %s\n", pcap_geterr(handle));
//                 printf("The counter was %ld\n", counterPing);
//             }

//             if(result == -1) 
//                 break;

//             totalCounter += counterPing;
//             totalPacketSize += packetSizePing;

//             if(verbose) printf("[PING] - %ld Packets Read From Pcap File\n", counterPing);
//         } else {
//             counterPing = 0;
//             packetSizePing = 0;

//             if(verbose) printf("[PING] - End Of Pcap File\n");

//             break;
//         } 

//         //pong
//         if(result != -2) {

//             if (Configfields::readPacketMode.compare("offline") == 0)                // TODO 
//                 result = readPacketOfflineMode(h_packetsMetadataPong, h_packetsMemPoolPong, handle, &counterPong, &packetSizePong, &startTime);
            
//             if(result < 0 && result != -2 && verbose) {
//                 printf("Something went wrong in reading packets(%d)\n", result);
//                 printf("The Error was : %s\n", pcap_geterr(handle));
//                 printf("The counter was %ld\n", counterPong);
//             }

//             if(result == -1) 
//                 break;

//             totalCounter += counterPong;
//             totalPacketSize += packetSizePong;

//             if(verbose) printf("[PONG] - %ld Packets Read From Pcap File\n", counterPing);
//         } else {
//             counterPong = 0;
//             packetSizePong = 0;

//             if(verbose) printf("[PONG] - End Of Pcap File\n");
//         }

//         //pang
//         if(result != -2) {

//             if (Configfields::readPacketMode.compare("offline") == 0)                // TODO 
//                 result = readPacketOfflineMode(h_packetsMetadataPang, h_packetsMemPoolPang, handle, &counterPang, &packetSizePang, &startTime);
            
//             if(result < 0 && result != -2 && verbose) {
//                 printf("Something went wrong in reading packets(%d)\n", result);
//                 printf("The Error was : %s\n", pcap_geterr(handle));
//                 printf("The counter was %ld\n", counterPang);
//             }

//             if(result == -1) 
//                 break;

//             totalCounter += counterPang;
//             totalPacketSize += packetSizePang;

//             if(verbose) printf("[PANG] - %ld Packets Read From Pcap File\n", counterPang);
//         } else {
//             counterPang = 0;
//             packetSizePang = 0;

//             if(verbose) printf("[PANG] - End Of Pcap File\n");
//         }

//         timerChunk.start();

//         // if (Configfields::isTimerSet) timerHDPing.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMemPoolPing, (void*) h_packetsMemPoolPing, packetSizePing, cudaMemcpyHostToDevice, pingStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMetadataPing, (void*) h_packetsMetadataPing, counterPing * sizeof(PacketMetadata), cudaMemcpyHostToDevice, pingStream));
//         // if (Configfields::isTimerSet) timerHDPing.end();
//         // if (Configfields::isTimerSet) durationHDPing = timerHDPing.elapsed();
//         // if (Configfields::isTimerSet) totalHDDuration += durationHDPing;

//         // if(verbose) printf("[PING] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Host To Device \n", (counterPing), (packetSizePing)/(_GB_), (counterPing * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PING]\t| DurationHDPing : %lf ms\n", durationHDPing);
//         // if(verbose && Configfields::isTimerSet) printf("[PING]\t| BandwidthHDPing : %lf Gb/s\n", ((packetSizePing + counterPing * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationHDPing));

//         // if (Configfields::isTimerSet) timerHDPong.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMemPoolPong, (void*) h_packetsMemPoolPong, packetSizePong, cudaMemcpyHostToDevice, pongStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMetadataPong, (void*) h_packetsMetadataPong, counterPong * sizeof(PacketMetadata), cudaMemcpyHostToDevice, pongStream));
//         // if (Configfields::isTimerSet) timerHDPong.end();
//         // if (Configfields::isTimerSet) durationHDPong = timerHDPong.elapsed();
//         // if (Configfields::isTimerSet) totalHDDuration += durationHDPong;

//         // if(verbose) printf("[PONG] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Host To Device \n", (counterPong), (packetSizePong)/(_GB_), (counterPong * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PONG]\t| DurationHDPong : %lf ms\n", durationHDPong);
//         // if(verbose && Configfields::isTimerSet) printf("[PONG]\t| BandwidthHDPong : %lf Gb/s\n", ((packetSizePong + counterPong * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationHDPong));

//         // if (Configfields::isTimerSet) timerHDPang.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMemPoolPang, (void*) h_packetsMemPoolPang, packetSizePang, cudaMemcpyHostToDevice, pangStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) d_packetsMetadataPang, (void*) h_packetsMetadataPang, counterPang * sizeof(PacketMetadata), cudaMemcpyHostToDevice, pangStream));
//         // if (Configfields::isTimerSet) timerHDPang.end();
//         // if (Configfields::isTimerSet) durationHDPang = timerHDPang.elapsed();
//         // if (Configfields::isTimerSet) totalHDDuration += durationHDPang;

//         // if(verbose) printf("[PANG] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Host To Device \n", (counterPang), (packetSizePang)/(_GB_), (counterPang * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PANG]\t| DurationHDPang : %lf ms\n", durationHDPang);
//         // if(verbose && Configfields::isTimerSet) printf("[PANG]\t| BandwidthHDPang : %lf Gb/s\n", ((packetSizePang + counterPang * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationHDPang));

//         HD_MODULE(timerHDPing, h_packetsMetadataPing, d_packetsMetadataPing, h_packetsMemPoolPing, d_packetsMemPoolPing, counterPing, pingStream, durationHDPing, totalHDDuration, verbose, packetSizePing, "PING");
//         HD_MODULE(timerHDPong, h_packetsMetadataPong, d_packetsMetadataPong, h_packetsMemPoolPong, d_packetsMemPoolPong, counterPong, pongStream, durationHDPong, totalHDDuration, verbose, packetSizePong, "PONG");
//         HD_MODULE(timerHDPang, h_packetsMetadataPang, d_packetsMetadataPang, h_packetsMemPoolPang, d_packetsMemPoolPang, counterPang, pangStream, durationHDPang, totalHDDuration, verbose, packetSizePang, "PANG");
        
//         if(verbose) printf("______________________________________________________________________\n");

//         totalHDPacketSize += packetSizePing + packetSizePong + packetSizePang + (counterPing + counterPong + counterPang) * sizeof(PacketMetadata);

//         /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        
//         int threadPerBlock = Configfields::threadPerBlock;
        
//         // if (Configfields::isTimerSet) timerKernelPing.start();
//         // performProcess<<<((counterPing + threadPerBlock - 1)/threadPerBlock), threadPerBlock, 0, pingStream>>>(d_packetsMetadataPing, d_packetsMemPoolPing, counterPing, d_trie);
//         // if (Configfields::isTimerSet) timerKernelPing.end();
//         // if (Configfields::isTimerSet) durationKernelPing = timerKernelPing.elapsed();
//         // if (Configfields::isTimerSet) totalKernelDuration += durationKernelPing;
                
//         // if(verbose) printf("[PING] -  RuleGraph Was Processed For %d Threads Per Block\n", threadPerBlock);
//         // if(verbose) printf("[PING] - %ld Packets (%.3lf GB) Processed On GPU\n", counterPing, ((packetSizePing) * 1.0)/(_GB_));
//         // if(verbose  && Configfields::isTimerSet) printf("[PING]\t| DurationKernelPing : %lf ms\n", durationKernelPing);
//         // if(verbose  && Configfields::isTimerSet) printf("[PING]\t| BandwidthKernelPing : %lf Gb/s\n", ((packetSizePing) * 1000.0 * 8.0)/(_GB_*durationKernelPing));

//         // if (Configfields::isTimerSet) timerKernelPong.start();
//         // performProcess<<<((counterPong + threadPerBlock - 1)/threadPerBlock), threadPerBlock, 0, pongStream>>>(d_packetsMetadataPong, d_packetsMemPoolPong, counterPong, d_trie);
//         // if (Configfields::isTimerSet) timerKernelPong.end();
//         // if (Configfields::isTimerSet) durationKernelPong = timerKernelPong.elapsed();
//         // if (Configfields::isTimerSet) totalKernelDuration += durationKernelPong;

//         // if(verbose) printf("[PONG] - RuleGraph Was Processed For %d Threads Per Block\n", threadPerBlock);
//         // if(verbose) printf("[PONG] - %ld Packets (%.3lf GB) Processed On GPU \n", counterPong, ((packetSizePong) * 1.0)/(_GB_));
//         // if(verbose  && Configfields::isTimerSet) printf("[PONG]\t| DurationKernelPong : %lf ms\n", durationKernelPong);
//         // if(verbose  && Configfields::isTimerSet) printf("[PONG]\t| BandwidthKernelPong : %lf Gb/s\n", ((packetSizePong) * 1000.0 * 8.0)/(_GB_*durationKernelPong));

//         // if (Configfields::isTimerSet) timerKernelPang.start();
//         // performProcess<<<((counterPang + threadPerBlock - 1)/threadPerBlock), threadPerBlock, 0, pangStream>>>(d_packetsMetadataPang, d_packetsMemPoolPang, counterPang, d_trie);
//         // if (Configfields::isTimerSet) timerKernelPang.end();
//         // if (Configfields::isTimerSet) durationKernelPang = timerKernelPang.elapsed();
//         // if (Configfields::isTimerSet) totalKernelDuration += durationKernelPang;

//         // if(verbose) printf("[PANG] - RuleGraph Was Processed For %d Threads Per Block\n", threadPerBlock);
//         // if(verbose) printf("[PANG] - %ld Packets (%.3lf GB) Processed On GPU \n", counterPang, ((packetSizePang) * 1.0)/(_GB_));
//         // if(verbose  && Configfields::isTimerSet) printf("[PANG]\t| DurationKernelPang : %lf ms\n", durationKernelPang);
//         // if(verbose  && Configfields::isTimerSet) printf("[PANG]\t| BandwidthKernelPang : %lf Gb/s\n", ((packetSizePang) * 1000.0 * 8.0)/(_GB_*durationKernelPang));

//         KERNEL_MODULE(timerKernelPing, counterPing, threadPerBlock, pingStream, d_packetsMetadataPing, d_packetsMemPoolPing, d_trie, durationKernelPing, totalKernelDuration, verbose, packetSizePing, "PING");
//         KERNEL_MODULE(timerKernelPong, counterPong, threadPerBlock, pongStream, d_packetsMetadataPong, d_packetsMemPoolPong, d_trie, durationKernelPong, totalKernelDuration, verbose, packetSizePong, "PONG");
//         KERNEL_MODULE(timerKernelPang, counterPang, threadPerBlock, pangStream, d_packetsMetadataPang, d_packetsMemPoolPang, d_trie, durationKernelPang, totalKernelDuration, verbose, packetSizePang, "PANG");

//         if(verbose) printf("______________________________________________________________________\n");

//         ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//         // if (Configfields::isTimerSet) timerDHPing.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMetadataPing, (void*) d_packetsMetadataPing, counterPing * sizeof(PacketMetadata), cudaMemcpyDeviceToHost, pingStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMemPoolPing, (void*) d_packetsMemPoolPing, packetSizePing, cudaMemcpyDeviceToHost, pingStream));
//         // if (Configfields::isTimerSet) timerDHPing.end();
//         // if (Configfields::isTimerSet) durationDHPing = timerDHPing.elapsed();
//         // if (Configfields::isTimerSet) totalDHDuration += durationDHPing;

//         // if(verbose) printf("[PING] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Device to Host\n", (counterPing), (packetSizePing)/(_GB_), (counterPing * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PING]\t| DurationDHPing : %lf ms\n", durationDHPing);
//         // if(verbose && Configfields::isTimerSet) printf("[PING]\t| BandwidthDHPing : %lf Gb/s\n", ((packetSizePing + counterPing * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationDHPing));

//         // if (Configfields::isTimerSet) timerDHPong.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMetadataPong, (void*) d_packetsMetadataPong, counterPong * sizeof(PacketMetadata), cudaMemcpyDeviceToHost, pongStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMemPoolPong, (void*) d_packetsMemPoolPong, packetSizePong, cudaMemcpyDeviceToHost, pongStream));
//         // if (Configfields::isTimerSet) timerDHPong.end();
//         // if (Configfields::isTimerSet) durationDHPong = timerDHPong.elapsed();
//         // if (Configfields::isTimerSet) totalDHDuration += durationDHPong;

//         // if(verbose) printf("[PONG] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Device to Host\n", (counterPong), (packetSizePong)/(_GB_), (counterPong * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PONG]\t| DurationDHPong : %lf ms\n", durationDHPong);
//         // if(verbose && Configfields::isTimerSet) printf("[PONG]\t| BandwidthDHPong : %lf Gb/s\n", ((packetSizePong + counterPong * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationDHPong));

//         // if (Configfields::isTimerSet) timerDHPang.start();
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMetadataPang, (void*) d_packetsMetadataPang, counterPang * sizeof(PacketMetadata), cudaMemcpyDeviceToHost, pangStream));
//         // CHECK_CUDA_ERROR(cudaMemcpyAsync((void*) h_packetsMemPoolPang, (void*) d_packetsMemPoolPang, packetSizePang, cudaMemcpyDeviceToHost, pangStream));
//         // if (Configfields::isTimerSet) timerDHPang.end();
//         // if (Configfields::isTimerSet) durationDHPang = timerDHPang.elapsed();
//         // if (Configfields::isTimerSet) totalDHDuration += durationDHPang;

//         // if(verbose) printf("[PANG] - %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered From Device to Host\n", (counterPang), (packetSizePang)/(_GB_), (counterPang * sizeof(PacketMetadata)) / (_GB_));
//         // if(verbose && Configfields::isTimerSet) printf("[PANG]\t| DurationDHPang : %lf ms\n", durationDHPang);
//         // if(verbose && Configfields::isTimerSet) printf("[PANG]\t| BandwidthDHPang : %lf Gb/s\n", ((packetSizePang + counterPang * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_*durationDHPang));

//         DH_MODULE(timerDHPing, h_packetsMetadataPing, d_packetsMetadataPing, h_packetsMemPoolPing, d_packetsMemPoolPing, counterPing, pingStream, durationDHPing, totalDHDuration, verbose, packetSizePing, "PING");
//         DH_MODULE(timerDHPong, h_packetsMetadataPong, d_packetsMetadataPong, h_packetsMemPoolPong, d_packetsMemPoolPong, counterPong, pongStream, durationDHPong, totalDHDuration, verbose, packetSizePong, "PONG");
//         DH_MODULE(timerDHPang, h_packetsMetadataPang, d_packetsMetadataPang, h_packetsMemPoolPang, d_packetsMemPoolPang, counterPang, pangStream, durationDHPang, totalDHDuration, verbose, packetSizePang, "PANG");

//         if(verbose) printf("______________________________________________________________________\n");

//         totalDHPacketSize += (packetSizePing + packetSizePong + packetSizePang) + (counterPing + counterPong + counterPang) * sizeof(PacketMetadata);

//         timerChunk.end();
//         durationChunk = timerChunk.elapsed();
//         totalDuration += durationChunk;

//         if(verbose) printf(">> %ld Packets (%lf GB Mempool and %lf GB Metadata) Transfered\n", (counterPing + counterPong + counterPang), (packetSizePing + packetSizePong + packetSizePang)/(_GB_), ((counterPing + counterPong + counterPang) * sizeof(PacketMetadata)) / (_GB_));
//         if(verbose) printf("\t| DurationChunk : %lf ms\n", durationChunk);
//         if(verbose) printf("\t| BandwidthChunk : %lf Gb/s\n", (((packetSizePing + packetSizePong + packetSizePang) + (counterPing + counterPong + counterPang) * sizeof(PacketMetadata)) * 1000.0 * 8.0)/(_GB_ * durationChunk));
//         if(verbose) printf("########################################################################\n\n");

//         cudaStreamSynchronize(pingStream);
//         cudaStreamSynchronize(pongStream);
//         cudaStreamSynchronize(pangStream);

//         //ping
//         for(size_t i = 0 ; i < counterPing ; i++) {  
//             PacketMetadata md = h_packetsMetadataPing[i];
//             PacketInfo* info;

//             ALIGN_ADDRESS(h_packetsMemPoolPing + md.packetOffset, PacketInfo, info);
//             ruleCount[info->ruleId]++;
//         }

//         //pong
//         for(size_t i = 0 ; i < counterPong ; i++) {  
//             PacketMetadata md = h_packetsMetadataPong[i];
//             PacketInfo* info;

//             ALIGN_ADDRESS(h_packetsMemPoolPong + md.packetOffset, PacketInfo, info);
//             ruleCount[info->ruleId]++;
//         }

//         //pang
//         for(size_t i = 0 ; i < counterPang ; i++) {  
//             PacketMetadata md = h_packetsMetadataPang[i];
//             PacketInfo* info;

//             ALIGN_ADDRESS(h_packetsMemPoolPang + md.packetOffset, PacketInfo, info);
//             ruleCount[info->ruleId]++;
//         }

//         if(!verbose){
//             printf("\033[2K\r");
//             fflush(stdout);

//             printf("# %0.3lf%% Of %s Is Processed", (((totalCounter*(16-sizeof(PacketInfo)) + totalPacketSize)*1.0)/(pcapFileSize*1.0))*100, pcapFilePath);
//             fflush(stdout);
//         } 

//         chunkCounter++;

//         if(result == -2)
//             break;
//     } printf("\n");
//     pcap_close(handle);


//     printf(">> Result:\n\t| Total Packets: %ld\n", totalCounter);
//     for(size_t i = 0 ; i < Rule_Count ; i++)
//         if(ruleCount[i] != 0) printf("\t| %s : %d\n", getRuleName(i), ruleCount[i]);
    
//     printf("\n\t| Duration: %lf ms\n\t| Bandwidth: %lf Gb/s\n\t| Bandwidth: %lf MPacket/s\n\t| Size: %lf Gb\n", 
//         totalDuration, ((totalPacketSize + totalCounter*sizeof(PacketInfo)) * 8.0 * 1000.0) / (totalDuration * _GB_), (totalCounter * 1000.0) / (totalDuration * _MB_)  ,(totalPacketSize * 8.0)/(_GB_));    
    
//     fprintf(resultFd, "%lf,%ld,%ld\n", ((totalPacketSize + totalCounter*sizeof(PacketInfo)) * 8.0 * 1000.0) / (totalDuration * _GB_), PACKET_BUFFER_CHUNK_SIZE, Configfields::threadPerBlock);

//     CHECK_CUDA_ERROR(cudaFree((void*) d_trie));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMemPoolPing));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMemPoolPong));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMemPoolPang));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMetadataPing));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMetadataPong));
//     CHECK_CUDA_ERROR(cudaFree((void*) d_packetsMetadataPang));

//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMemPoolPing));
//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMemPoolPong));
//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMemPoolPang));
//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMetadataPing));
//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMetadataPong));
//     CHECK_CUDA_ERROR(cudaFreeHost((void*) h_packetsMetadataPang));

//     CHECK_CUDA_ERROR(cudaStreamDestroy(pingStream));
//     CHECK_CUDA_ERROR(cudaStreamDestroy(pongStream));
//     CHECK_CUDA_ERROR(cudaStreamDestroy(pangStream));

//     fclose(resultFd);

//     return 0;
// }


// static int processDirectory(const char* directoryPath, bool verbose) {
//     struct dirent* entry;
//     DIR* dp;

//     dp = opendir(directoryPath);
//     if(dp == NULL) {
//         printf("Unable To Open Directory %s\n", directoryPath);
//         return -1;
//     }

//     while((entry = readdir(dp)) != NULL) {
//         if(entry->d_type == DT_REG && hasPcapExtension(entry->d_name)) {
//             char fullPath[1024];
//             snprintf(fullPath, sizeof(fullPath), "%s/%s", directoryPath, entry->d_name);            
//             processPcapFile(fullPath, verbose);
//         }
//     }

//     closedir(dp);
//     return 0;
// }

// #define HELP_COMMAND_LINE       "Usage: ./ruleGraph [options] <arguments>"                                                  \
//                                 "\nAvailable Options:"                                                                      \
//                                 "\n\t-f\t\t: Select The Pcap File"                                                          \
//                                 "\n\t-d\t\t: Select The Directory Containing Multiple Pcap Files"                           \
//                                 "\n\t-c\t\t: Select The Config File (Default Is Set To 'config.yml')"                       \
//                                 "\n\t-v\t\t: Make The Operation More Talkative"                                             \
//                                 "\n\t-h\t\t: Print Help And Exit\n" 

// int main(int argc, char* argv[]) {

//     if(argc == 1) {
//         printf(HELP_COMMAND_LINE);
//         return -1;
//     }

//     int opt;
//     bool processDir = false;
//     bool processFile = false;
//     bool haveConfigFileName = false;
//     bool verbose = false;
//     char* pstr = NULL;
//     char* configFilePath = NULL;

//     while((opt = getopt(argc, argv, "d:f:c:hv")) != -1) {
//         switch (opt)
//         {
//         case 'd':
//             processDir = true;
//             pstr = optarg;
//             break;

//         case 'f':
//             processFile = true;
//             pstr = optarg;
//             break;

//         case 'c':
//             haveConfigFileName = true;
//             configFilePath = optarg;
//             break;

//         case 'h':   
//             printf(HELP_COMMAND_LINE);
//             return 0;

//         case 'v':
//             verbose = true;
//             break;

//         case ':':
//             printf("Option -%c requires an argument\n", optopt);
//             return -1;

//         case '?':
//             printf("Unknown Option: -%c\n", optopt);
//             return -1;
//         }
//     }

//     if (!haveConfigFileName) {
//         configFilePath = "config.yml";  // default
//     }

//     ConfigLoader::loadAllfields(configFilePath);

//     if(processDir) 
//         return processDirectory(pstr, verbose);

//     if(processFile)
//         return processPcapFile(pstr, verbose);


//     return -1;
// }