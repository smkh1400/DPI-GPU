#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pcap.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <endian.h>
#include <time.h>

#include "rulesGraph.cuh"
#include "header.h"

#define CHECK_CUDA_ERROR(fun)                                                   \
{                                                                               \
    hipError_t err = fun;                                                      \
    if(err != hipSuccess) {                                                    \
        printf("CUDA at %s:%d: %s\n", __FUNCTION__, __LINE__ , hipGetErrorString(err));           \
        return -1;                                                               \
    }                                                                           \
}

#define swapEndian16(x)     ((uint16_t) (((x) >> 8) | ((x) << 8)))

#if __BYTE_ORDER == __LITTLE_ENDIAN
    #define htons(x) swapEndian16(x)
    #define ntohs(x) swapEndian16(x)
#else 
    #define htons(x) x
    #define ntohs(x) x
#endif

__global__ static void registerRuleGraph(InspectorNode* nodes) {
    InspectorNode* ethr_insp = &nodes[0];
    ethr_insp->setInspectorFunction((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;
        out.checkConditionResult = true;

        EthrHeader* hdr = (EthrHeader*) (p->getHeaderData());
        out.extractedCondition = &(hdr->ethrType);

        out.calculatedOffset = sizeof(EthrHeader);

        return out;
    });

    InspectorNode* ethrArp_insp = &nodes[1];
    ethrArp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint16_t ethrType = *((uint16_t*) cond);
        out.checkConditionResult = (ethrType == htons(0x0806));

        out.extractedCondition = NULL;

        out.calculatedOffset = 0;

        return out;
    }, Rule_EthrArp);


    InspectorNode* ethrIpv4_insp = &nodes[2];
    ethrIpv4_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint16_t ethrType = *((uint16_t*) cond);
        out.checkConditionResult = (ethrType == htons(0x0800));

        IPv4Header* hdr = (IPv4Header*) (p->getHeaderData());
        out.extractedCondition = &(hdr->protocol);

        size_t optionSize = (hdr->ihl*4)-20;
        out.calculatedOffset = sizeof(IPv4Header) + optionSize;

        return out;
    }, Rule_EthrIpv4);

    InspectorNode* ethrIpv4Icmp_insp = &nodes[3];
    ethrIpv4Icmp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint8_t protocol = *((uint8_t*) cond);
        out.checkConditionResult = (protocol == 0x01);

        out.extractedCondition = NULL;

        out.calculatedOffset = sizeof(ICMPHeader);

        return out;
    }, Rule_EthrIPv4ICMP);

    InspectorNode* ethrIpv4Udp_insp = &nodes[4];
    ethrIpv4Udp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint8_t protocol = *((uint8_t*) cond);
        out.checkConditionResult = (protocol == 0x11);

        UDPHeader* hdr = (UDPHeader*) (p->getHeaderData());
        out.extractedCondition = &(hdr->sport);

        out.calculatedOffset = sizeof(UDPHeader);

        return out;
    }, Rule_EthrIpv4Udp);

    InspectorNode* ethrIpv4UdpDns_insp = &nodes[5];
    ethrIpv4UdpDns_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint16_t sport = *((uint16_t*) cond);
        uint16_t dport = *((uint16_t*) (cond+2));
        out.checkConditionResult = ((sport == htons(0x0035)) || (dport == htons(0x0035)));

        out.extractedCondition = NULL;

        out.calculatedOffset = sizeof(DNSHeader);

        return out;
    }, Rule_EthrIpv4UdpDns);

    InspectorNode* ethrIpv4Tcp_insp = &nodes[6];
    ethrIpv4Tcp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint8_t protocol = *((uint8_t*) cond);
        out.checkConditionResult = (protocol == 0x06);

        TCPHeader* hdr = (TCPHeader*) (p->getHeaderData());
        int headerLength = hdr->headerLength * 4;
        out.extractedCondition = (p->getHeaderData() + headerLength);

        out.calculatedOffset = headerLength;

        return out;
    }, Rule_EthrIpv4Tcp);

    InspectorNode* ethrIpv4TcpHttp_insp = &nodes[7];
    ethrIpv4TcpHttp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint8_t* method = (uint8_t*) cond;
        out.checkConditionResult = 
            (method[0]=='G' && method[1]=='E' && method[2]=='T') ||
            (method[0]=='P' && method[1]=='O' && method[2]=='S' && method[3]=='T') ||
            (method[0]=='P' && method[1]=='U' && method[2]=='T') ||
            (method[0]=='D' && method[1]=='E' && method[2]=='L' && method[3]=='E' && method[4]=='T' && method[5]=='E') ||
            (method[0]=='H' && method[1]=='E' && method[2]=='A' && method[3]=='D') ||
            (method[0]=='O' && method[1]=='P' && method[2]=='T' && method[3]=='I' && method[4]=='O' && method[5]=='N' && method[6]=='S') ||
            (method[0]=='P' && method[1]=='A' && method[2]=='T' && method[3]=='C' && method[4]=='H') ||
            (method[0]=='T' && method[1]=='R' && method[2]=='A' && method[3]=='C' && method[4]=='E') ||
            (method[0]=='C' && method[1]=='O' && method[2]=='N' && method[3]=='N' && method[4]=='E' && method[5]=='C' && method[6]=='T');

        out.extractedCondition = NULL;

        out.calculatedOffset = 0;

        return out;
    }, Rule_EthrIpv4TcpHttp);    

    InspectorNode* ethrIpv4UdpRtp_insp = &nodes[8];
    ethrIpv4UdpRtp_insp->setRule((Inspector_t) [](HeaderBuffer* p, void* cond) -> InspectorFuncOutput {
        InspectorFuncOutput out;

        uint8_t version = *((uint8_t*) p->getHeaderData());
        out.checkConditionResult = (version >> 6 == 0b10) && (p->packetLen >= (p->headerOffset + 12));

        out.calculatedOffset = 0;

        out.extractedCondition = NULL;

        return out;
    }, Rule_EthrIpv4UdpRtp);


    
    ethr_insp->addChild(ethrArp_insp);
    ethr_insp->addChild(ethrIpv4_insp);
        ethrIpv4_insp->addChild(ethrIpv4Icmp_insp);
        ethrIpv4_insp->addChild(ethrIpv4Udp_insp);
            ethrIpv4Udp_insp->addChild(ethrIpv4UdpDns_insp);
            // ethrIpv4Udp_insp->addChild(ethrIpv4UdpRtp_insp);
        ethrIpv4_insp->addChild(ethrIpv4Tcp_insp);
            ethrIpv4Tcp_insp->addChild(ethrIpv4TcpHttp_insp);
}

__global__ void performProcess(PacketBuffer* packets, size_t packetCount, InspectorNode* rootNode) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    HeaderBuffer h;

    if(idx < packetCount) {
        for(size_t i = 0 ; i < HEADER_BUFFER_DATA_MAX_SIZE ; i++)
            h.headerData[i] = packets[idx].packetData[i];
        
        h.packetLen = packets[idx].packetLen;

        rootNode[0].processNode(&h, NULL);
        packets[idx].ruleId.clone(h.ruleId);
    }
        
}


int main() {
    size_t packetCount = 200000000;
    unsigned long long packetSize = 0L;
    PacketBuffer* packets = (PacketBuffer*) calloc(packetCount, sizeof(PacketBuffer));
    if(packets == NULL) 
        printf("Unable to allocate Packets\n");
    long long counter = 0L;

    {
        char errBuf[PCAP_ERRBUF_SIZE];
        pcap_t* handle;
        const u_char *packet;
        struct pcap_pkthdr* header = (struct pcap_pkthdr*) malloc(sizeof(*header));

        // handle = pcap_open_offline("/home/hamed/hpc_event_2/problem_big/cbig.pcap", errBuf);
        handle = pcap_open_offline("../3.pcap", errBuf);
        if(handle == NULL) {
            printf("Unable To Open Pcap File\n");
            return 1;
        }

        int result;
        while((result = (pcap_next_ex(handle, &header, &packet))) >= 0) {
            PacketBuffer p(packet, header->caplen);
            packets[counter++] = p;

            packetCount++;
            packetSize += header->caplen;
            if ((counter*sizeof(PacketBuffer)) >= (24696061952L)) break;        // GPU RAM Limit
        } 

        pcap_close(handle);
    }

    packetCount = counter;

    float duration = 0;
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    printf("Packets Was Read From Pcap File (%lf GB)\n", (packetSize*1.0)/(1024.0 * 1024.0 * 1024.0));

    PacketBuffer* d_packets;
    CHECK_CUDA_ERROR(hipMalloc((void**) &d_packets, packetCount*sizeof(PacketBuffer)));

    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy((void*) d_packets, (void*) packets, packetCount*sizeof(PacketBuffer), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(stop));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&duration, start, stop));

    printf("*** Packets (%lf GB) Transfered From Host To Device ***\n", (packetCount*sizeof(PacketBuffer))/(1024.0*1024.0*1024.0));
    printf("\t Duration : %lf ms\n", duration);
    printf("\t Bandwidth : %lf Gb/s\n", (packetCount*sizeof(PacketBuffer)*1000.0*8.0)/(1024.0*1024.0*1024.0*duration));
    printf("\t Bandwidth : %lf GPacket/s\n\n", (packetCount*1000.0)/(1024.0*1024.0*1024.0*duration));


    InspectorNode* d_nodes;
    CHECK_CUDA_ERROR(hipMalloc((void**) &d_nodes, 9*sizeof(InspectorNode)));

    registerRuleGraph<<<1,1>>>(d_nodes);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    printf("RuleGraph Was Registered On Device\n");

    int threadPerBlock = 512;

    CHECK_CUDA_ERROR(hipEventRecord(start));
    performProcess<<<(packetCount+threadPerBlock-1)/threadPerBlock,threadPerBlock>>>(d_packets, packetCount, d_nodes);
    CHECK_CUDA_ERROR(hipEventRecord(stop));

    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&duration, start, stop));

    printf("*** RuleGraph Was Processed For %d Threads Per Block ***\n", threadPerBlock);
    printf("**** %ld Packets (%.3lf GB) Process On GPU ***\n", packetCount, (packetSize*1.0)/(1024.0*1024.0*1024.0));
    printf("\t Duration : %lf ms\n", duration);
    printf("\t Bandwidth : %lf Gb/s\n", (packetSize*1000.0*8.0)/(1024.0*1024.0*1024.0*duration));
    printf("\t Bandwidth : %lf GPacket/s\n\n", (packetCount*1000.0)/(1024.0*1024.0*1024.0*duration));

    CHECK_CUDA_ERROR(hipMemcpy((void*) packets, (void*) d_packets, packetCount*sizeof(PacketBuffer), hipMemcpyDeviceToHost));

    int ruleCount[Rule_Count] = {0};
    for(size_t i = 0 ; i < packetCount ; i++)
        for(int j = 0 ; j < QUEUE_BUFFER_MAX_LEN ; j++)
            ruleCount[packets[i].ruleId.array[j]]++;

    for(size_t i = 0 ; i < Rule_Count ; i++)
        printf("#%s : %d\n", getRuleName(i), ruleCount[i]);
    

    return 0;
}